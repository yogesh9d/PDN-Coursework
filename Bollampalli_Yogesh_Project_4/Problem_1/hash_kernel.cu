
#include <hip/hip_runtime.h>

/* Hash Kernel --------------------------------------
*       Generates an array of hash values from nonces.
*/
#define MAX     123123123

__global__
void hash_kernel(unsigned int* hash_array, unsigned int* nonce_array, unsigned int array_size, unsigned int* transactions, unsigned int n_transactions, unsigned int mod) {

    // Calculate thread index
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

    // TODO: Generate hash values
    if (index < array_size){
        hash_array[index] = (nonce_array[index] + transactions[0] * (index + 1)) % MAX;
        for (int j = 1; j < n_transactions; j++)
            hash_array[index] = (hash_array[index] + transactions[j] * (index + 1)) % MAX;
    }

} // End Hash Kernel //
