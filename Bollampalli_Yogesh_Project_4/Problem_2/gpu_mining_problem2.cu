#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "support.h"
#include "hash_kernel.cu"
#include "nonce_kernel.cu"
#include "reduction_kernel.cu"


// to activate debug statements
#define DEBUG 1

// program constants
#define BLOCK_SIZE 1024
#define SEED       123

// solution constants
#define MAX     123123123
#define TARGET  20

// functions used
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int* transactions, unsigned int n_transactions);
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions);
void err_check(hipError_t ret, char* msg, int exit_code);


/* Main ------------------ //
*   This is the main program.
*/
int main(int argc, char* argv[]) {

    // Catch console errors
    if (argc != 6) {
        printf("USE LIKE THIS: gpu_mining transactions.csv n_transactions trials out.csv time.csv\n");
        return EXIT_FAILURE;
    }


    // Output files
    FILE* output_file = fopen(argv[4], "w");
    FILE* time_file   = fopen(argv[5], "w");

    // Read in the transactions
    unsigned int n_transactions = strtoul(argv[2], NULL, 10);
    unsigned int* transactions = (unsigned int*)calloc(n_transactions, sizeof(unsigned int));
    read_file(argv[1], transactions, n_transactions);

    // get the number of trials
    unsigned int trials = strtoul(argv[3], NULL, 10);


    // -------- Start Mining ------------------------------------------------------- //
    // ----------------------------------------------------------------------------- //

    // Set timer and cuda error return
    Timer timer;
    startTime(&timer);
    hipError_t cuda_ret;

    // To use with kernels
    int count_blocks = ceil((float)trials / (float)BLOCK_SIZE);
    dim3 dimGrid(count_blocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);


    // ------ Step 1: generate the nonce values ------ //

    // Allocate the nonce device memory
    unsigned int* device_nonce_array;
    cuda_ret = hipMalloc((void**)&device_nonce_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate nonces to device memory!", 1);

    // Launch the nonce kernel
    nonce_kernel <<< dimGrid, dimBlock >>> (
        device_nonce_array, // put nonces into here
        trials,             // size of array
        MAX,                // to mod with
        SEED                // random seed
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch nonce kernel!", 2);

    // Get nonces from device memory
    unsigned int* nonce_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read nonce from device memory!", 3);


    // ------ Step 2: Generate the hash values ------ //

    // TODO Problem 1: perform this hash generation in the GPU
    // Hint: You need both nonces and transactions to compute a hash.

    // Allocating the hash device memory
    unsigned int* device_hash_array;
    cuda_ret = hipMalloc((void**)&device_hash_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate hashes to device memory!", 4);

     // Allocating the transaction device memory
    unsigned int* device_transactions;
    cuda_ret = hipMalloc((void**)&device_transactions, n_transactions * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate transactions to device memory!", 5);

     // Copying transactions from host to device memory
    cuda_ret = hipMemcpy(device_transactions, transactions, n_transactions * sizeof(unsigned int), hipMemcpyHostToDevice);
    err_check(cuda_ret, (char*)"Unable to read transactions from host memory!", 6);

    // Launch the nonce kernel
    hash_kernel<<<dimGrid, dimBlock>>>(
        device_hash_array,  // hash array on device
        device_nonce_array, // nonce array on device
        trials,             // size of hash and nonce arrays
        device_transactions,// transactions array on device
        n_transactions,     // number of transactions
        MAX                 // mod value to mod with
    );

    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch hash kernel!", 7);

    // Get hashes from device memory
    unsigned int* hash_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read hashes from device memory!", 8);

    // Free memory
    free(transactions);


    // ------ Step 3: Find the nonce with the minimum hash value ------ //

    // TODO Problem 2: find the minimum in the GPU by reduction

    // Launch the reduction kernel
    reduction_kernel<<<dimGrid, dimBlock>>>(
        device_hash_array,  // hash array on device
        device_nonce_array, // nonce array on device
        trials              // size of hash and nonce arrays
    );

    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch reduction kernel!", 9);

    // Get hashes from device memory
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read hash from device memory!", 10);

    // Get nonces from device memory
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read nonce from device memory!", 11);

    unsigned int min_hash  = MAX;
    unsigned int min_nonce = MAX;
    for (int i = 0; i < count_blocks; i++) {
        if (hash_array[i] < min_hash) {
            min_hash  = hash_array[i];
            min_nonce = nonce_array[i];
        }
    }

    // Free memory
    free(nonce_array);
    free(hash_array);

    stopTime(&timer);
    // ----------------------------------------------------------------------------- //
    // -------- Finish Mining ------------------------------------------------------ //


    // Get if suceeded
    char* res = (char*)malloc(8 * sizeof(char));
    if (min_hash < TARGET)  res = (char*)"Success!";
    else                    res = (char*)"Failure.";

    // Show results in console
    if (DEBUG)
        printf("%s\n   Min hash:  %u\n   Min nonce: %u\n   %f seconds\n",
            res,
            min_hash,
            min_nonce,
            elapsedTime(timer)
        );

    // Print results
    fprintf(output_file, "%s\n%u\n%u\n", res, min_hash, min_nonce);
    fprintf(time_file, "%f\n", elapsedTime(timer));

    // Cleanup
    fclose(time_file);
    fclose(output_file);

    return 0;
} // End Main -------------------------------------------- //



/* Generate Hash ----------------------------------------- //
*   Generates a hash value from a nonce and transaction list.
*/
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int* transactions, unsigned int n_transactions) {

    unsigned int hash = (nonce + transactions[0] * (index + 1)) % MAX;
    for (int j = 1; j < n_transactions; j++) {
        hash = (hash + transactions[j] * (index + 1)) % MAX;
    }
    return hash;

} // End Generate Hash ---------- //



/* Read File -------------------- //
*   Reads in a file of transactions.
*/
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions) {

    // open file
    FILE* trans_file = fopen(file, "r");
    if (trans_file == NULL)
        fprintf(stderr, "ERROR: could not read the transaction file.\n"),
        exit(-1);

    // read items
    char line[100] = { 0 };
    for (int i = 0; i < n_transactions && fgets(line, 100, trans_file); ++i) {
        char* p;
        transactions[i] = strtof(line, &p);
    }

    fclose(trans_file);

} // End Read File ------------- //



/* Error Check ----------------- //
*   Exits if there is a CUDA error.
*/
void err_check(hipError_t ret, char* msg, int exit_code) {
    if (ret != hipSuccess)
        fprintf(stderr, "%s \"%s\".\n", msg, hipGetErrorString(ret)),
        exit(exit_code);
} // End Error Check ----------- //
